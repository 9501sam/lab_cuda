#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <fstream>

//TODO: CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    //hint: implement the matrix multiplication
}

void matrixMultiplyGPU(float* h_A, float* h_B, float* h_C, int N) {
    // The matrix size
    size_t size = N * N * sizeof(float);

    //TODO: Allocate device(GPU) memory
    //hint: cudaMalloc(??, ??)
    float *d_A, *d_B, *d_C;

    //TODO: Copy matrices from host(CPU) to device(GPU)
    //hint: cudaMemcpy(??, ??, ??, ??)
    

    //TODO: Define block and grid dimensions
    //hint: dim3 threadsPerBlock(??, ??)
    //hint: dim3 numBlocks(??, ??)

    //TODO: Launch the matrix multiplication kernel
    //hint: matrixMulKernel<<<??, ??>>>(?, ?, ?, ?)

    //TODO: Copy result from device to host
    //hint: cudaMemcpy(??, ??, ??, ??);
    

    //TODO: Free device memory
    //hint: cudaFree()
}

int main() {
    int N = 2000; // Example size of the matrix
    size_t size = N * N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize matrices with some values
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i);
    }

    // Perform matrix multiplication
    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiplyGPU(h_A, h_B, h_C, N);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;

    // Write the last row of the result matrix to a file
    std::ofstream outputFile("gpu_output.txt");
    if (outputFile.is_open()) {
        outputFile << h_C[0] << " ";
        outputFile << "\n";
        outputFile.close();
    } else {
        std::cerr << "Unable to open file for writing\n";
    }

    std::cout << "finish" << std::endl;
    std::cout << "GPU Matrix multiplication took " << duration.count() << " ms\n";

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}


// nvc++ matrix_mul_gpu.cu -o matrix_mul_gpu
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <fstream>

//TODO: CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    //hint: implement the matrix multiplication
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];

        }
        C[row * N + col] = sum;

    }
}

void matrixMultiplyGPU(float* h_A, float* h_B, float* h_C, int N) {
    // The matrix size
    size_t size = N * N * sizeof(float);

    //TODO: Allocate device(GPU) memory
    //hint: cudaMalloc(??, ??)
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    //TODO: Copy matrices from host(CPU) to device(GPU)
    //hint: cudaMemcpy(??, ??, ??, ??)
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    //TODO: Define block and grid dimensions
    //hint: dim3 threadsPerBlock(??, ??)
    //hint: dim3 numBlocks(??, ??)
    int blockSize = 8; // Choose a block size, e.g., 16x16
    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 numBlocks((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    //TODO: Launch the matrix multiplication kernel
    //hint: matrixMulKernel<<<??, ??>>>(?, ?, ?, ?)
    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    //TODO: Copy result from device to host
    //hint: cudaMemcpy(??, ??, ??, ??);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    //TODO: Free device memory
    //hint: cudaFree()
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 2000; // Example size of the matrix
    size_t size = N * N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize matrices with some values
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i);
    }

    // Perform matrix multiplication
    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiplyGPU(h_A, h_B, h_C, N);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;

    // Write the last row of the result matrix to a file
    std::ofstream outputFile("gpu_output.txt");
    if (outputFile.is_open()) {
        outputFile << h_C[0] << " ";
        outputFile << "\n";
        outputFile.close();
    } else {
        std::cerr << "Unable to open file for writing\n";
    }

    std::cout << "finish" << std::endl;
    std::cout << "GPU Matrix multiplication took " << duration.count() << " ms\n";

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}


// nvc++ matrix_mul_gpu.cu -o matrix_mul_gpu
